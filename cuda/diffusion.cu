#include "hip/hip_runtime.h"
//kernel definition
__global__ void diffusionSolver(double* A, double * old,int n_x,int n_y)
{

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if(i*(n_x-i-1)*j*(n_y-j-1)!=0)
        A[i+n_y*j] = A[i+n_y*j] + (old[i-1+n_y*j]+old[i+1+n_y*j]+
                       old[i+(j-1)*n_y]+old[i+(j+1)*n_y] -4*old[i+n_y*j])/40;


}

int main()
{


    int i,j ,M;
    M = n_y ;
    phi = (double *) malloc( n_x*n_y* sizeof(double));
    phi_old = (double *) malloc( n_x*n_y* sizeof(double));
    dummy = (double *) malloc( n_x*n_y* sizeof(double));
    int iterationMax =10;
    //phase initialization
    for(j=0;j<n_y ;j++)
    {
        for(i=0;i<n_x;i++)
        {
            if((.4*n_x-i)*(.6*n_x-i)<0)
                phi[i+M*j] = -1;
            else
                phi[i+M*j] = 1;

            phi_old[i+M*j] = phi[i+M*j];
        }
    }

    double *dev_phi;
    hipMalloc((void **) &dev_phi, n_x*n_y*sizeof(double));

    dim3 threadsPerBlock(100,10);
    dim3 numBlocks(n_x*n_y / threadsPerBlock.x, n_x*n_y / threadsPerBlock.y);

    //start iterating
    for(int z=0; z<iterationMax; z++)
    {
        //copy array on host to device
        hipMemcpy(dev_phi, phi, n_x*n_y*sizeof(double),
                hipMemcpyHostToDevice);

        //call kernel
        diffusionSolver<<<numBlocks, threadsPerBlock>>>(dev_phi, phi_old,n_x,n_y);

        //get updated array back on host
        hipMemcpy(phi, dev_phi,n_x*n_y*sizeof(double), hipMemcpyDeviceToHost);

        //old values will be assigned new values
        for(j=0;j<n_y ;j++)
        {
            for(i=0;i<n_x;i++)
            {
                phi_old[i+n_y*j] = phi[i+n_y*j];
            }
        }
    }

    return 0;
}
